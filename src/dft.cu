#include "hip/hip_runtime.h"
#include "dft.h"

using namespace std;

__global__ void dftKernel(cuda::std::complex<double> *dftResult, unsigned long sigLength, double *timeSignal, int tasksPerThread)
{
    auto group = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();

    // printf("group %d", group.thread_rank());

    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // unsigned long tid = group.thread_rank();
    // printf("tid %d\n", tid);
    // cuda::std::complex<double>* holding = (cuda::std::complex<double>*) malloc(tasksPerThread * sizeof(cuda::std::complex<double>));

    cuda::std::complex<double> j(0.0, 1.0);

    double pi = 2*acos(0.0);

    for (int tid = tasksPerThread * threadId; tid < tasksPerThread * threadId + tasksPerThread; tid++)
    {
        cuda::std::complex<double> compSum = 0;
        for (int nIdx = 0; nIdx < sigLength; nIdx++)
        {
            double fractFactor = (double) (nIdx * tid) / (double) sigLength;
            compSum += timeSignal[nIdx] * exp(-2 * pi * j * fractFactor);
        }
        dftResult[tid] = compSum;
    }
}

// https://pages.di.unipi.it/gemignani/woerner.pdf
complex<double>* dft::iterative(double *timeSignal, unsigned long sigLength)
{
    cout << "Running Iterative DFT" << endl;

    double pi = 2*acos(0.0);

    int numRounds = (int) log2(sigLength);

    complex<double>* dftResult = (complex<double>*) malloc(sigLength * sizeof(complex<double>));
    
    // auto start = chrono::high_resolution_clock::now();

    complex<double> j(0.0, 1.0);

    for (int kIdx = 0; kIdx < sigLength; kIdx++)
    {
        complex<double> compSum = 0;
        for (int nIdx = 0; nIdx < sigLength; nIdx++)
        {
            double fractFactor = (double) (nIdx * kIdx) / (double) sigLength;
            compSum += timeSignal[nIdx] * exp(-2 * pi * j * fractFactor);
        }
        dftResult[kIdx] = compSum;
        // cout << compSum << endl;
    }


    // auto stop = chrono::high_resolution_clock::now();
    // auto diff = chrono::duration_cast<chrono::microseconds>(stop - start);
    // cout << "iterative algo time (us): " << diff.count() << endl;

    return dftResult;

}

// Binary Exchange algorithm for parallel FFT
complex<double>* dft::cudaParallel(double *timeSignal, unsigned long sigLength)
{
    cout << "Running CUDA Parallelized DFT" << endl;

    complex<double>* dftResult = (complex<double>*) malloc(sigLength * sizeof(complex<double>));

    int threadsPerBlock;
    int numBlocks;
    int tasksPerThread;
    if (sigLength <= 1024)
    {
        threadsPerBlock = sigLength;
        numBlocks = 1;
        tasksPerThread = 1;
    }
    else
    {
        threadsPerBlock = 1024;
        numBlocks = (int) ceil(sigLength / 1024.0);
        tasksPerThread = 1;
    }

    // Can not spawn more than 64 blocks (probably more but will need to be power of 2 and could not spawn 128 blocks)
    if (numBlocks > 64)
    {
        numBlocks = 64;
        tasksPerThread = (int) ceil(sigLength / (1024.0 * 64.0));
    }

    double pi = 2*acos(0.0);

    cuda::std::complex<double>* d_dftResult;
    double* d_timeSignal;

    hipMalloc((void**) &d_dftResult, sizeof(cuda::std::complex<double>) * (int) sigLength);
    hipMalloc((void**) &d_timeSignal, sizeof(double) * (int) sigLength);

    hipMemcpy(d_timeSignal, timeSignal, sizeof(double) * sigLength, hipMemcpyHostToDevice);

    // auto start = chrono::high_resolution_clock::now();

    // cout << "round " << roundIdx << endl;

    void *kernelArgs[] = { &d_dftResult, &sigLength, &d_timeSignal, &tasksPerThread};
    // int dev = 0;
    // hipDeviceProp_t deviceProp;
    // hipGetDeviceProperties(&deviceProp, dev);
    // initialize, then launch

    dim3 dimBlock(threadsPerBlock, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    hipLaunchCooperativeKernel((void*)dftKernel, dimGrid, dimBlock, kernelArgs);
    // hipDeviceSynchronize();
    // binEx<<<1, sigLength>>>(d_bitShufTimeSig, sigLength, roundIdx, tasksPerThread, d_tempHold);
    
    // auto stop = chrono::high_resolution_clock::now();
    // auto diff = chrono::duration_cast<chrono::microseconds>(stop - start);

    // cout << "parallel algo time (us): " << diff.count() << endl;

    hipMemcpy(dftResult, d_dftResult, sizeof(cuda::std::complex<double>) * sigLength, hipMemcpyDeviceToHost);
    hipFree(d_dftResult);
    hipFree(d_timeSignal);

    return dftResult;

}

complex<double>* dft::ompParallel(double *timeSignal, unsigned long sigLength)
{
    cout << "Running OMP Parallelized DFT" << endl;

    complex<double>* dftResult = (complex<double>*) malloc(sigLength * sizeof(complex<double>));

    double pi = 2*acos(0.0);

    int numThreads = omp_get_max_threads() - 1;
    
    // auto start = chrono::high_resolution_clock::now();

    complex<double> j(0.0, 1.0);
    
    #pragma omp parallel for num_threads(numThreads)
    for (int kIdx = 0; kIdx < sigLength; kIdx++)
    {
        complex<double> compSum = 0;
        for (int nIdx = 0; nIdx < sigLength; nIdx++)
        {
            double fractFactor = (double) (nIdx * kIdx) / (double) sigLength;
            compSum += timeSignal[nIdx] * exp(-2 * pi * j * fractFactor);
        }
        dftResult[kIdx] = compSum;
    }

    // auto stop = chrono::high_resolution_clock::now();
    // auto diff = chrono::duration_cast<chrono::microseconds>(stop - start);
    // cout << "omp algo time (us): " << diff.count() << endl;

    return dftResult;

}
